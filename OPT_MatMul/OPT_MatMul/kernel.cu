#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <direct.h>
#include <io.h>
#include <windows.h>
#include <math.h>
#include <processthreadsapi.h>
#include "hip/hip_runtime.h"
#include ""
float *ImportMatrix(char *filename, int *N, int *M)
{
	FILE *handle;
	float *val;
	int i, j;
	printf("Reading File %s\n", filename);
	if (filename == NULL)
	{
		return NULL;
	}

	handle = fopen(filename, "r");
	if (handle == NULL)
	{
		printf("Error opening file: %s\n", strerror(errno));
		return NULL;
	}
	fscanf(handle, "%d %d", N, M);
	val = (float *)malloc(*N * *M * sizeof(float));
	for (i = 0; i < *N; i++)
	{
		for (j = 0; j < *M; j++)
		{
			fscanf(handle, "%f ", val + (i * *M) + j);
		}
	}
	fclose(handle);
	return val;
}
int ExportMatrix(char *filename, float *val, int N, int M)
{
	FILE *handle;
	int i, j;
	printf("Writing File %s\n", filename);
	if (filename == NULL)
	{
		return NULL;
	}

	handle = fopen(filename, "w");
	if (handle == NULL)
	{
		printf("Failed to open %s\n", filename);
		return NULL;
	}
	fprintf(handle, "%d %d\n", N, M);
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < M; j++)
		{
			fprintf(handle, "%.2f ", *(val + (i * M) + j));
		}
		fprintf(handle, "\n");
	}
	fclose(handle);
	return 1;
}

#define WIDTH = 16
// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C, int numARows,
									 int numAColumns, int numBRows, int numBColumns)
{
	// TODO: Insert code to implement matrix multiplication here

	__shared__ float A_[WIDTH][WIDTH];
	__shared__ float B_[WIDTH][WIDTH];

	int tx = threadIdx.x,
		ty = threadIdx.y,
		bx = blockIdx.x,
		by = blockIdx.y;

	int row = blockIdx.y * WIDTH + threadIdx.y;
	int col = blockIdx.x * WIDTH + threadIdx.x;

	float result = 0;

	for (int i = 0; i < (numAColumns / WIDTH) + 1; i++)
	{
		if (row < numARows && i * WIDTH + threadIdx.x < numAColumns)
		{
			A_[ty][tx] = A[row * numAColumns + i * WIDTH + tx];
		}
		else
		{
			A_[ty][tx] = 0;
		}

		if (col < numBColumns && i * WIDTH + ty < numBRows)
		{
			B_[ty][tx] = B[(i * WIDTH + ty) * numBColumns + col];
		}
		else
		{
			B_[ty][tx] = 0;
		}

		__syncthreads();

		for (int j = 0; j < WIDTH; j++)
		{
			result += A_[ty][j] * B[j][tx];
		}
		__syncthreads();
	}
}

LARGE_INTEGER Time_start()
{
	LARGE_INTEGER StartingTime;
	QueryPerformanceCounter(&StartingTime);
	return StartingTime;
}
int Elapsed_time(LARGE_INTEGER StartingTime, const char *message, int prt)
{
	LARGE_INTEGER EndingTime, Frequency, ElapsedMicroseconds;
	QueryPerformanceFrequency(&Frequency);
	QueryPerformanceCounter(&EndingTime);
	ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart;
	ElapsedMicroseconds.QuadPart *= 1000000;
	ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;
	if (prt == 1)
	{
		printf("%s Elapsed Time %lld in micro-seconds\n", message, ElapsedMicroseconds.QuadPart);
	}
	return 0;
}

int main(int argc, char **argv)
{

	float *hostA; // The A matrix
	float *hostB; // The B matrix
	float *hostC; // The output C matrix
	float *hostD; // Solution D Matrix
	float *deviceA;
	float *deviceB;
	float *deviceC;
	int numARows;	 // number of rows in the matrix A
	int numAColumns; // number of columns in the matrix A
	int numBRows;	 // number of rows in the matrix B
	int numBColumns; // number of columns in the matrix B
	int numCRows;
	int numCColumns;
	LARGE_INTEGER StartingTime;
	int i, j, prt = 0;
	float meanDiff = 0;

	if (argc != 6)
	{
		printf("GPU_MatMul Expected_Out InFile1 InFile2 myOutFile prt\n");
		return NULL;
	}

	printf("Running GPU Matrix Multiplicaion V1.2...\n");
	if (strcmp(argv[5], "ON") == 0)
		prt = 1;

	StartingTime = Time_start();
	hostD = (float *)ImportMatrix(argv[1], &numCRows, &numCColumns);
	hostA = (float *)ImportMatrix(argv[2], &numARows, &numAColumns);
	hostB = (float *)ImportMatrix(argv[3], &numBRows, &numBColumns);
	/* This code must be deleted*/

	Elapsed_time(StartingTime, "Reading Data.", prt);

	// Allocate the hostC matrix

	StartingTime = Time_start();
	numCRows = numARows;
	numCColumns = numBColumns;

	hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));

	// Should be deleted once your code works

	for (i = 0; i < numCRows; i++)
	{
		for (j = 0; j < numCColumns; j++)
		{
			hostC[i * numCColumns + j] = hostD[i * numCColumns + j];
		}
	}

	Elapsed_time(StartingTime, "Allocating GPU memory.", prt);

	printf("The dimensions of A are is %d x %d \n", numARows, numAColumns);
	printf("The dimensions of B are is %d x %d \n", numBRows, numBColumns);
	printf("The dimensions of C are is %d x %d \n", numCRows, numCColumns);

	StartingTime = Time_start();
	// TODO: Allocate GPU memory here
	hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
	hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
	hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));

	Elapsed_time(StartingTime, "Allocating GPU memory.", prt);

	StartingTime = Time_start();
	// TODO: Copy memory to the GPU here
	hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);

	Elapsed_time(StartingTime, "Copying input memory to the GPU.", prt);

	// TODO: Initialize the grid and block dimensions here
	// Here you will have to use dim3
	dim3 blockDim((numCColumns / WIDTH) + 1, (numCRows / WIDTH) + 1);
	dim3 gridDim(WIDTH, WIDTH);

	StartingTime = Time_start();
	// TODO:: Launch the GPU Kernel here

	hipDeviceSynchronize();
	matrixMultiply<<<blockDim, gridDim>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, );

	Elapsed_time(StartingTime, "--------->Performing CUDA computation*******", prt);

	StartingTime = Time_start();
	// TODO:: Copy the GPU memory back to the CPU here

	hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);
	Elapsed_time(StartingTime, "Copying output memory to the CPU.", prt);

	StartingTime = Time_start();
	// TODO:: Free the GPU memory here
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	Elapsed_time(StartingTime, "Freeing GPU Memory.", prt);

	ExportMatrix(argv[4], hostC, numCRows, numCColumns);
	/*Perform Success Test*/

	for (i = 0; i < numCRows; i++)
	{
		for (j = 0; j < numCColumns; j++)
		{
			meanDiff = meanDiff + fabs(hostC[i * numCColumns + j] - hostD[i * numCColumns + j]);
		}
	}
	meanDiff = meanDiff / (float)(numBColumns * numARows);
	if (meanDiff > 0.01)
	{
		printf("%f Failed\n", meanDiff);
	}
	else
	{
		printf("Passed\n");
	}

	free(hostA);
	free(hostB);
	free(hostC);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
